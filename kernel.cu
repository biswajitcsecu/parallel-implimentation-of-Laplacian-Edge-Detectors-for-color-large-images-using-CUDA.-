#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32
#define RAD 1

int divUp(int a, int b) { return (a + b - 1) / b; }

__device__
float clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__device__
int idxClip(int idx, int idxMax) {
return idx > (idxMax-1) ? (idxMax-1) : (idx < 0 ? 0 : idx);
}

__device__
int flatten(int col, int row, int width, int height) {
return idxClip(col, width) + idxClip(row, height)*width;
}

__global__
void sharpenKernel(float4 *d_out, const float4 *d_in,const float *d_filter, int w, int h) {

const int c = threadIdx.x + blockDim.x * blockIdx.x;
const int r = threadIdx.y + blockDim.y * blockIdx.y;

if ((c >= w) || (r >= h)) return;
  const int i = flatten(c, r, w, h);
  const int fltSz = 2*RAD + 1;
  float rgb[3] = {0.f, 0.f, 0.f};

for (int rd = -RAD; rd <= RAD; ++rd) {
  for (int cd = -RAD; cd <= RAD; ++cd) {
    int imgIdx = flatten(c + cd, r + rd, w, h);
    int fltIdx = flatten(RAD + cd, RAD + rd, fltSz, fltSz);
    float4 color = d_in[imgIdx];
    float weight = d_filter[fltIdx];
    rgb[0] += weight*color.x;
    rgb[1] += weight*color.y;
    rgb[2] += weight*color.z;
  }
}

d_out[i].x = clip(rgb[0]);
d_out[i].y = clip(rgb[1]);
d_out[i].z = clip(rgb[2]);

}

void sharpenParallel(float4 *arr, int w, int h) {

const int fltSz = 2 * RAD + 1;

const float filter[9] = {-0.5, 1.0, 0.5,
			1.0, -4.0, 1.0,
			0.5, 1.0, -0.5};

float4 *d_in = 0, *d_out = 0;
float *d_filter = 0;

hipMalloc(&d_in, w*h*sizeof(float4));

hipMemcpy(d_in, arr, w*h*sizeof(float4), hipMemcpyHostToDevice);

hipMalloc(&d_out, w*h*sizeof(float4));

hipMalloc(&d_filter, fltSz*fltSz*sizeof(float));

hipMemcpy(d_filter, filter, fltSz*fltSz*sizeof(float),hipMemcpyHostToDevice);

const dim3 blockSize(TX, TY);
const dim3 gridSize(divUp(w, blockSize.x), divUp(h, blockSize.y));

sharpenKernel<<<gridSize, blockSize>>>(d_out, d_in, d_filter, w, h);

hipMemcpy(arr, d_out, w*h*sizeof(float4), hipMemcpyDeviceToHost);

hipFree(d_in);
hipFree(d_out);
hipFree(d_filter);

}














